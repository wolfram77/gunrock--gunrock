#include "hip/hip_runtime.h"
#include <gunrock/algorithms/pr.hxx>
#include <gunrock/util/performance.hxx>
#include <gunrock/io/parameters.hxx>
#include <cmath>
#include <vector>
#include <omp.h>

using namespace gunrock;
using namespace memory;




/**
 * Compute the L1-norm of the difference of two arrays in parallel.
 * @param x an array
 * @param y another array
 * @param N size of arrays
 * @param a initial value
 * @returns ||x-y||_1
 */
template <class TX, class TY, class TA=TX>
inline TA l1NormDeltaOmp(const TX *x, const TY *y, size_t N, TA a=TA()) {
  // ASSERT(x && y);
  #pragma omp parallel for schedule(auto) reduction(+:a)
  for (size_t i=0; i<N; ++i)
    a += TA(std::abs(x[i] - y[i]));
  return a;
}




void test_pr(int num_arguments, char** argument_array) {
  // --
  // Define types

  using vertex_t = int;
  using edge_t = int;
  using weight_t = float;

  using csr_t =
      format::csr_t<memory_space_t::device, vertex_t, edge_t, weight_t>;

  // --
  // IO

  gunrock::io::cli::parameters_t params(num_arguments, argument_array,
                                        "Page Rank");

  printf("Loading graph %s ...\n", params.filename.c_str());
  io::matrix_market_t<vertex_t, edge_t, weight_t> mm;
  auto [properties, coo] = mm.load(params.filename);
  printf("order: %d, size: %d, symmetric: %d\n", coo.number_of_rows, coo.number_of_nonzeros, properties.symmetric);

  csr_t csr;

  if (params.binary) {
    csr.read_binary(params.filename);
  } else {
    csr.from_coo(coo);
  }

  // --
  // Build graph

  auto G = graph::build<memory_space_t::device>(properties, csr);

  // --
  // Params and memory allocation

  srand(time(NULL));

  weight_t alpha = 0.85;
  weight_t tol = 1e-10;

  size_t n_vertices = G.get_number_of_vertices();
  size_t n_edges = G.get_number_of_edges();
  thrust::device_vector<weight_t> p(n_vertices);

  // Parse tags
  std::vector<std::string> tag_vect;
  gunrock::io::cli::parse_tag_string(params.tag_string, &tag_vect);

  // --
  // GPU Run

  std::vector<float> run_times;
  printf("Running PR ...\n");
  auto benchmark_metrics =
      std::vector<benchmark::host_benchmark_t>(params.num_runs);
  for (int i = 0; i < params.num_runs; i++) {
    benchmark::INIT_BENCH();

    run_times.push_back(gunrock::pr::run(G, alpha, tol, p.data().get()));

    benchmark::host_benchmark_t metrics = benchmark::EXTRACT();
    benchmark_metrics[i] = metrics;

    benchmark::DESTROY_BENCH();
  }

  // Placeholder since PR does not use sources
  std::vector<int> src_placeholder;

  // Export metrics
  if (params.export_metrics) {
    gunrock::util::stats::export_performance_stats(
        benchmark_metrics, n_edges, n_vertices, run_times, "pr",
        params.filename, "market", params.json_dir, params.json_file,
        src_placeholder, tag_vect, num_arguments, argument_array);
  }

  // Log
  print::head(p, 40, "GPU rank");

  // Copy p to host
  thrust::host_vector<weight_t> p_host = p;

  // Run PR with zero tolerance to find exact PageRanks
  printf("Running exact PR ...\n");
  thrust::device_vector<weight_t> p_exact(n_vertices);
  gunrock::pr::run(G, alpha, 0, p_exact.data().get());
  thrust::host_vector<weight_t> p_exact_host = p_exact;

  // Compute L1-norm of the difference between exact and approximate PageRanks
  printf("Computing error ...\n");
  weight_t l1_norm = l1NormDeltaOmp(p_host.data(), p_exact_host.data(), n_vertices);

  std::cout << "GPU Elapsed Time : " << run_times[params.num_runs - 1]
            << " (ms)" << std::endl;
  std::cout << "GPU Error : " << l1_norm << std::endl;
}

int main(int argc, char** argv) {
  test_pr(argc, argv);
}
